#include "hip/hip_runtime.h"
%%writefile Assign4B.cu

#include<iostream>
#include<chrono>
#include<hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;


__global__ void mulPar(int * A, int * B, int * C, int M,int N, int K){
    int row=blockIdx.y*blockDim.y + threadIdx.y;
    int col=blockIdx.x*blockDim.x+ threadIdx.x;

    if(row<M && col<K){
        int sum=0;
        for(int i=0; i<N; i++){
            sum+=A[row*N+i]+B[i*K+col];
        }
        C[row*K+col]=sum;
    }

}
void initialize(int * matrix, int rows, int cols){
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            cout<<"Enter the element at "<<i<<" "<<j<<" : "; cin>>matrix[i*cols+j];
        }
    }
}

void print(int * matrix, int rows, int cols){
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
          cout<<matrix[i*cols+j]<<" ";
        }
        cout<<endl;
    }
    cout<<endl;
}

void mulSeq(int * A, int * B, int * C, int M,int N, int K){
    for(int i=0; i<M; i++){
        for(int j=0; j<K; j++){
            int sum=0;
            for(int k=0; k<N; k++){
                sum+=A[i*N+k]*B[k*K+j];
            }
            C[i*K+j]=sum;
        }
    }
}


int main(){
    int M,N,K;
    cout<<"Enter the MxNxK : "; cin>>M>>N>>K;

    int * A,*B,*C;

    int matsize=M*K;
    size_t matBytes=matsize*sizeof(int);

    A=new int[M*N];
    B=new int[N*K];
    C=new int[M*K];

    initialize(A,M,N);
    initialize(B,N,K);

    print(A,M,N);
    print(B,N,K);

    int *X,*Y,*Z;

    hipMalloc(&X,M*N*sizeof(int));
    hipMalloc(&Y,N*K*sizeof(int));
    hipMalloc(&Z,M*K*sizeof(int));

    hipMemcpy(X,A,M*N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(Y,B,N*K*sizeof(int),hipMemcpyHostToDevice);

    int Threads=16;
    int Blocks=(M+Threads-1)/Threads;

    dim3 threads(Threads,Threads);
    dim3 blocks(Blocks,Blocks);


    // multi seq

    auto start=high_resolution_clock::now();
    mulSeq(A,B,C,M,N,K);
    auto stop=high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop-start);

    cout<<"Mul matrix : "<<endl;
    print(C,M,K);
    cout<<"Duration for seq : "<<duration.count()<<" microseconds"<<endl;


    // mul par
    start=high_resolution_clock::now();
    mulPar<<<blocks, threads>>>(X,Y,Z,M,N,K);
    hipMemcpy(C,Z,matBytes,hipMemcpyDeviceToHost);
     stop=high_resolution_clock::now();
     duration = duration_cast<microseconds>(stop-start);

    cout<<"Mul matrix : "<<endl;
    print(C,M,K);
    cout<<"Duration for par : "<<duration.count()<<" microseconds"<<endl;

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);
    return 0;
}

// !nvcc filename -o fileobj 
// ! ./fileobj 