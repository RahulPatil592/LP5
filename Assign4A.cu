// %%writefile Assign4A.cu

#include<iostream>
#include<chrono>
#include<hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

__global__ void add(int * A, int * B, int * C, int size){
    int tid=blockIdx.x * blockDim.x + threadIdx.x;

    if(tid<size){
        C[tid]=A[tid]+B[tid];
    }
}

void initialize(int * vector,int size){
    for(int i=0; i<size; i++){
        cout<<"Enter element "<<i+1<<" for vector : "; cin>>vector[i];
    }
}

void print(int * vector, int size){
    for(int i=0; i<size; i++){
        cout<<vector[i]<<" ";
    }
    cout<<endl;
}

void addSequential(int * A, int * B, int * C, int size){
    for(int i=0; i<size; i++){
        C[i]=A[i]+B[i];
    }
}

int main(){

    int N;
    cout<<"Enter the size of the vectors : "; cin>>N;

    int* A,*B,*C;

    int vsize=N;

    size_t vectorBytes=vsize*sizeof(int);

    A=new int[vsize];
    B=new int[vsize];
    C=new int[vsize];

    initialize(A,vsize);
    initialize(B,vsize);

    cout<<"Vector A : "; 
    print(A,vsize);
    cout<<"Vector B : ";
    print(B,vsize);

    int *X,*Y,*Z;
    hipMalloc(&X,vectorBytes);
    hipMalloc(&Y,vectorBytes);
    hipMalloc(&Z,vectorBytes);

    hipMemcpy(X,A,vectorBytes,hipMemcpyHostToDevice);
    hipMemcpy(Y,B,vectorBytes,hipMemcpyHostToDevice);

    int threadsPerBlock=256;
    int blocksPerGrid=(N+threadsPerBlock-1)/threadsPerBlock;

    //seq add
    auto start=high_resolution_clock::now();
    addSequential(A,B,C,N);
    auto stop=high_resolution_clock::now();
    auto duration=duration_cast<microseconds>(stop-start);

    cout<<"Sequential addition : ";
    print(C,N);

    cout<<"Sequential addition time : "<<duration.count()<<" microseconds"<<endl;

    //parallel add
    start=high_resolution_clock::now();
    add<<<blocksPerGrid,threadsPerBlock>>>(X,Y,Z,N);
    hipMemcpy(C,Z,vectorBytes,hipMemcpyDeviceToHost);
    stop=high_resolution_clock::now();
    duration=duration_cast<microseconds>(stop-start);

    cout<<"Parallel addition : ";
    print(C,N);

    cout<<"Parallel addition time : "<<duration.count()<<" microseconds"<<endl;

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);
    return 0;
}

// !nvcc Assign4A.cu -o Assign4A
// ! ./Assign4A